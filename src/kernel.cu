#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include <iostream>

__global__ void kernel(int *a, int *b, int *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

void test_function()
{
    std::cout << "Starting CUDA" << std::endl;
    int a[1024], b[1024], c[1024];
    int *c_a, *c_b, *c_c;
    int n = 1024;
    int num_blocks = 256;

    std::cout << "Initializing arrays" << std::endl;
    hipMalloc(&c_a, n * sizeof(int));
    hipMalloc(&c_b, n * sizeof(int));
    hipMalloc(&c_c, n * sizeof(int));

    std::cout << "Filling values" << std::endl;
    for (int i = 0; i < n; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    std::cout << "Copying to device" << std::endl;
    hipMemcpy(c_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    std::cout << "Running kernel" << std::endl;
    kernel<<<num_blocks, n / num_blocks>>>(c_a, c_b, c_c, n);

    std::cout << "Copying to host" << std::endl;
    hipMemcpy(c, c_c, n * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Printing result" << std::endl;
    for (int i = 0; i < n; i++)
    {
        std::cout << c[i] << ", ";
    }
    std::cout << std::endl;

    std::cout << "Freeing memory" << std::endl;
    hipFree(c_a);
    hipFree(c_b);
    hipFree(c_c);
}
